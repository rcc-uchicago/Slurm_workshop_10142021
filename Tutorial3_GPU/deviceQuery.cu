#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


int main(int argc, char **argv){
    hipDeviceProp_t prop;
    int count;
    hipGetDeviceCount(&count);
    printf("Number of GPUs on the Node is %d\n", count);
    for (int i=0; i < count; ++i){
        hipGetDeviceProperties(&prop, i);
        printf("--- General Information for device %d----\n", i);
        printf("Name:  %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("GPU Clock rate: %.0f MHz\n", prop.clockRate*1e-3f);
        printf("------Memory Information for device %d\n", i);
        printf("Total global mem: %.2f GBytes\n", (float) prop.totalGlobalMem / pow(1024, 3));
        printf("Memory clock rate: %.0f MHz\n", prop.memoryClockRate * 1e-3f);
        printf("Memory bus width: %d-bit \n", prop.memoryBusWidth );
        printf("L2 cache size: %d Bytes\n", prop.l2CacheSize );
        
        printf("Threads and Blocks information for %d GPU");
        printf(" Maximum number of Threads per block %d\n", prop.maxThreadsPerBlock);
        printf(" Maximum size of each dimension of block %d x  %d x %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf(" Maximum sizes of each dimension of grid %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
 
        return 0;
    
    }






return 0;

}
