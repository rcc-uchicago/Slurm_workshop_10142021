#include <stdio.h>
#include <stdio.h>
#include <stddef.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>

extern "C" void listdev( int rank ){
    hipError_t err;
    int dev_cnt = 0;
    err = hipGetDeviceCount( &dev_cnt );
    assert( err == hipSuccess || err == hipErrorNoDevice );
    //printf( "rank %d, cnt %d\n", rank, dev_cnt );
    hipDeviceProp_t prop;
    for (int dev = 0; dev < dev_cnt; ++dev) {
        err = hipGetDeviceProperties( &prop, dev );
        assert( err == hipSuccess );
        printf( "rank %d, dev %d, prop %s, pci %d, %d, %d\n",
        rank, dev,
        prop.name,
        prop.pciBusID,
        prop.pciDeviceID,
        prop.pciDomainID );
        }   
}
